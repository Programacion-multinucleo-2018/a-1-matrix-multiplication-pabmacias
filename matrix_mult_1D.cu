#include "hip/hip_runtime.h"
#include "common.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <chrono>

using namespace std;

#define SIZE 2000;

void initialData(int *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (int)(rand());
    }

    return;
}

void printMatrix(int *A, const int nx, const int ny)
 {
     int *ia = A;

     for (int iy = 0; iy < ny; iy++)
     {
         for (int ix = 0; ix < nx; ix++)
         {
           printf("%d     ", ia[ix]);
         }
         printf("\n");
         ia += nx;
     }

     return;
 }

void multMatrixOnHost(int *A, int *B, int *C, const int nx,
                     const int ny)
{
    int *ia = A;
    int *ib = B;
    int *ic = C;

    int *ibm = B;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ibm = B;
            for (int im = 0; im < nx; im++)
            {
                ic[ix] += ia[im] * ibm[ix];
                ibm += nx;
            }
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}


void checkResult(int *hostRef, int *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %d gpu %d\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

// grid 1D block 1D
__global__ void multMatrixOnGPU1D(int *MatA, int *MatB, int *MatC, int nx,
                                 int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx )
        for (int iy = 0; iy < ny; iy++)
        {
            int idx = iy * nx + ix;

            for (int im = 0; im < ny; im++)
            {
              int idxm = iy * nx + im;
              int idym = im * nx + ix;
              MatC[idx] += MatA[idxm] * MatB[idym];
            }
        }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // set up data size of matrix
    int nx = SIZE;
    int ny = SIZE;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    int *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (int *)malloc(nBytes);
    h_B = (int *)malloc(nBytes);
    hostRef = (int *)malloc(nBytes);
    gpuRef = (int *)malloc(nBytes);

    // initialize data at host side

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    /* Deje comentada esta parte porque estaba tardando mucho en cpu para hacer
    las pruebas, pero si lo cheque antes y los resultados estaban bien */

    // add matrix at host side for result SAFE_CALLs
    /*auto start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    auto end_cpu =  chrono::high_resolution_clock::now();
    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("multMatrixOnHost elapsed %f ms\n", duration_ms.count());*/

    // malloc device global memory
    int *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side256
    int dimx = 128*2;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);

    auto start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnGPU1D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    auto end_cpu =  chrono::high_resolution_clock::now();

    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("multMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,
           grid.y,
           block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // check device results
    // checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
